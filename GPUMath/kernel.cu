#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <malloc.h>
#include <stdio.h>
#include <math.h>
constexpr auto NThreads = 1024;

__global__ void kernelMatrixAddMatrix(float* matrixA, float* matrixB, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrixA[i] + matrixB[i];
}

__global__ void kernelMatrixAddNum(float* matrix, float number, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrix[i] + number;
}

__global__ void kernelMatrixSubMatrix(float* matrixA, float* matrixB, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrixA[i] - matrixB[i];
}

__global__ void kernelMatrixSubNum(float* matrix, float number, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrix[i] - number;
}

__global__ void kernelMatrixMultMatrix(float* matrixA, float* matrixB, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrixA[i] * matrixB[i];
}

__global__ void kernelMatrixMultNum(float* matrix, float number, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrix[i] * number;
}

__global__ void kernelMatrixDivMatrix(float* matrixA, float* matrixB, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrixA[i] / matrixB[i];
}

__global__ void kernelMatrixDivNum(float* matrix, float number, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrix[i] / number;
}

__global__ void kernelMatrixScalerProduct(float* matrixA, float* matrixB, float* resultMatrix, int bColumnsCount, int dimension) {
	
	__shared__ float cache[NThreads];

	int i = blockIdx.x;
	int k = threadIdx.x;

	int x = i / bColumnsCount;
	int y = i % bColumnsCount;

	if (k < dimension) {
		cache[k] = matrixA[x * dimension + k] * matrixB[y + bColumnsCount * k];
	}
	__syncthreads();

	int n = 0;
	int iterations = log2((double)NThreads);
	int offset = NThreads / 2;

	while (n < iterations) {
		if (k < offset) {
			cache[k] += cache[offset + k];
			offset /= 2;
		}
		__syncthreads();
		n++;
	}

	resultMatrix[i] = cache[0];
}

extern "C" {
	__declspec(dllexport) void matrixAddMatrix(float* matrixA, float* matrixB, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrixA;
		float* dev_matrixB;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrixA, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_matrixB, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrixA, matrixA, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		hipMemcpy(dev_matrixB, matrixB, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixAddMatrix <<<elemsCount, NThreads >>> (dev_matrixA, dev_matrixB, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrixA);
		hipFree(dev_matrixB);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixAddNum(float* matrix, const float number, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrix;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrix, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrix, matrix, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixAddNum << <elemsCount, 1 >> > (dev_matrix, number, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrix);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixSubMatrix(float* matrixA, float* matrixB, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrixA;
		float* dev_matrixB;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrixA, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_matrixB, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrixA, matrixA, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		hipMemcpy(dev_matrixB, matrixB, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixSubMatrix << <elemsCount, 1 >> > (dev_matrixA, dev_matrixB, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrixA);
		hipFree(dev_matrixB);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixSubNum(float* matrix, const float number, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrix;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrix, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrix, matrix, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixSubNum <<<elemsCount, 1 >> > (dev_matrix, number, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrix);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixMultMatrix(float* matrixA, float* matrixB, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrixA;
		float* dev_matrixB;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrixA, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_matrixB, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrixA, matrixA, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		hipMemcpy(dev_matrixB, matrixB, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixMultMatrix << <elemsCount, 1 >> > (dev_matrixA, dev_matrixB, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrixA);
		hipFree(dev_matrixB);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixMultNum(float* matrix, const float number, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrix;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrix, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrix, matrix, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixMultNum << <elemsCount, 1 >> > (dev_matrix, number, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrix);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixDivMatrix(float* matrixA, float* matrixB, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrixA;
		float* dev_matrixB;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrixA, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_matrixB, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrixA, matrixA, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		hipMemcpy(dev_matrixB, matrixB, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixDivMatrix << <elemsCount, 1 >> > (dev_matrixA, dev_matrixB, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrixA);
		hipFree(dev_matrixB);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixDivNum(float* matrix, const float number, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrix;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrix, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrix, matrix, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixDivNum << <elemsCount, 1 >> > (dev_matrix, number, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrix);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixScalerProduct(float* matrixA, float* matrixB, float* resultMatrix, const int aRowsCount, const int bColumnsCount, const int dimension) {
		int elemsCount = aRowsCount * bColumnsCount;
		float* dev_matrixA;
		float* dev_matrixB;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrixA, sizeof(float) * aRowsCount * dimension);
		hipMalloc((void**)&dev_matrixB, sizeof(float) * bColumnsCount * dimension);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrixA, matrixA, sizeof(float) * aRowsCount * dimension, hipMemcpyHostToDevice);
		hipMemcpy(dev_matrixB, matrixB, sizeof(float) * bColumnsCount * dimension, hipMemcpyHostToDevice);
		//hipMemcpy(dev_resultMatrix, resultMatrix, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixScalerProduct << <elemsCount, dimension >> > (dev_matrixA, dev_matrixB, dev_resultMatrix, bColumnsCount, dimension);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrixA);
		hipFree(dev_matrixB);
		hipFree(dev_resultMatrix);
	}
}