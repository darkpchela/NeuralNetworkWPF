#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <malloc.h>
#include <stdio.h>

__global__ void kernelMatrixAddMatrix(float* matrixA, float* matrixB, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrixA[i] + matrixB[i];
}

__global__ void kernelMatrixAddNum(float* matrix, float number, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrix[i] + number;
}

__global__ void kernelMatrixSubMatrix(float* matrixA, float* matrixB, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrixA[i] - matrixB[i];
}

__global__ void kernelMatrixSubNum(float* matrix, float number, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrix[i] - number;
}

__global__ void kernelMatrixMultMatrix(float* matrixA, float* matrixB, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrixA[i] * matrixB[i];
}

__global__ void kernelMatrixMultNum(float* matrix, float number, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrix[i] * number;
}

__global__ void kernelMatrixDivMatrix(float* matrixA, float* matrixB, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrixA[i] / matrixB[i];
}

__global__ void kernelMatrixDivNum(float* matrix, float number, float* resultMatrix) {
	int i = blockIdx.x;
	resultMatrix[i] = matrix[i] / number;
}

__global__ void kernelMatrixScalerProduct(float* matrixA, float* matrixB, float* resultMatrix, int bColumnsCount, int dimension) {
	int i = blockIdx.x;

	int j = i / bColumnsCount;
	int l = i % bColumnsCount;

	for (int k = 0; k < dimension; k++)
	{
		resultMatrix[i] += matrixA[j * dimension + k] * matrixB[l + bColumnsCount * k];
	}
}

extern "C" {
	__declspec(dllexport) void matrixAddMatrix(float* matrixA, float* matrixB, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrixA;
		float* dev_matrixB;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrixA, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_matrixB, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrixA, matrixA, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		hipMemcpy(dev_matrixB, matrixB, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixAddMatrix << <elemsCount, 1 >> > (dev_matrixA, dev_matrixB, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrixA);
		hipFree(dev_matrixB);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixAddNum(float* matrix, const float number, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrix;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrix, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrix, matrix, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixAddNum << <elemsCount, 1 >> > (dev_matrix, number, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrix);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixSubMatrix(float* matrixA, float* matrixB, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrixA;
		float* dev_matrixB;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrixA, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_matrixB, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrixA, matrixA, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		hipMemcpy(dev_matrixB, matrixB, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixSubMatrix << <elemsCount, 1 >> > (dev_matrixA, dev_matrixB, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrixA);
		hipFree(dev_matrixB);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixSubNum(float* matrix, const float number, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrix;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrix, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrix, matrix, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixSubNum << <elemsCount, 1 >> > (dev_matrix, number, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrix);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixMultMatrix(float* matrixA, float* matrixB, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrixA;
		float* dev_matrixB;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrixA, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_matrixB, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrixA, matrixA, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		hipMemcpy(dev_matrixB, matrixB, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixMultMatrix << <elemsCount, 1 >> > (dev_matrixA, dev_matrixB, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrixA);
		hipFree(dev_matrixB);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixMultNum(float* matrix, const float number, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrix;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrix, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrix, matrix, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixMultNum << <elemsCount, 1 >> > (dev_matrix, number, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrix);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixDivMatrix(float* matrixA, float* matrixB, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrixA;
		float* dev_matrixB;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrixA, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_matrixB, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrixA, matrixA, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		hipMemcpy(dev_matrixB, matrixB, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixDivMatrix << <elemsCount, 1 >> > (dev_matrixA, dev_matrixB, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrixA);
		hipFree(dev_matrixB);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixDivNum(float* matrix, const float number, float* resultMatrix, const int rowsCount, const int columnsCount) {
		int elemsCount = rowsCount * columnsCount;
		float* dev_matrix;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrix, sizeof(float) * elemsCount);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrix, matrix, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixDivNum << <elemsCount, 1 >> > (dev_matrix, number, dev_resultMatrix);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrix);
		hipFree(dev_resultMatrix);
	}

	__declspec(dllexport) void matrixScalerProduct(float* matrixA, float* matrixB, float* resultMatrix, const int aRowsCount, const int bColumnsCount, const int dimension) {
		int elemsCount = aRowsCount * bColumnsCount;
		float* dev_matrixA;
		float* dev_matrixB;
		float* dev_resultMatrix;
		hipMalloc((void**)&dev_matrixA, sizeof(float) * aRowsCount * dimension);
		hipMalloc((void**)&dev_matrixB, sizeof(float) * bColumnsCount * dimension);
		hipMalloc((void**)&dev_resultMatrix, sizeof(float) * elemsCount);
		hipMemcpy(dev_matrixA, matrixA, sizeof(float) * aRowsCount * dimension, hipMemcpyHostToDevice);
		hipMemcpy(dev_matrixB, matrixB, sizeof(float) * bColumnsCount * dimension, hipMemcpyHostToDevice);
		hipMemcpy(dev_resultMatrix, resultMatrix, sizeof(float) * elemsCount, hipMemcpyHostToDevice);
		kernelMatrixScalerProduct << <elemsCount, 1 >> > (dev_matrixA, dev_matrixB, dev_resultMatrix, bColumnsCount, dimension);
		hipMemcpy(resultMatrix, dev_resultMatrix, sizeof(float) * elemsCount, hipMemcpyDeviceToHost);
		hipFree(dev_matrixA);
		hipFree(dev_matrixB);
		hipFree(dev_resultMatrix);
	}
}